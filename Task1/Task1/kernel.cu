#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""

#define N   10

__global__ void add(int *a, int *b, int *c)
{
	int tid = threadIdx.x;
	while (tid < N)
	{
		c[tid] = a[tid] + b[tid];
		//printf("%d + %d = %d\n", a[tid], b[tid], c[tid]);
		tid += 1;

	}
}

__global__ void Threadadd(int *a, int *b, int *c)
{
	int tid = threadIdx.x;
	c[tid] = a[tid] + b[tid];
	//printf("%d + %d = %d\n", a[tid], b[tid], c[tid]);
}

__global__ void Blockadd(int *a, int *b, int *c)
{
	int tid = blockIdx.x;
	c[tid] = a[tid] + b[tid];
	//printf("%d + %d = %d\n", a[tid], b[tid], c[tid]);
}

__host__ int main(void) 
{
	int a[N], b[N], c[N];

	for (int i = 0; i<N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	int* devA;
	int* devB;
	int* devC;

	hipMalloc((void**)&devA, sizeof(int) * N);
	hipMalloc((void**)&devB, sizeof(int) * N);
	hipMalloc((void**)&devC, sizeof(int) * N);

	hipMemcpy(devA, a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(devB, b, sizeof(int) * N, hipMemcpyHostToDevice);

	add <<<1, 1 >>> (devA, devB, devC);

	hipEvent_t syncEvent;

	hipEventCreate(&syncEvent);    
	hipEventRecord(syncEvent, 0);  
	hipEventSynchronize(syncEvent);  

	hipMemcpy(c, devC, sizeof(int) * N, hipMemcpyDeviceToHost);

	for (int i = 0; i<N; i++) 
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipEventDestroy(syncEvent);

	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	/////////////////////////////////////////
	int a1[N], b1[N], c1[N];

	for (int i = 0; i<N; i++)
	{
		a1[i] = i;
		b1[i] = i * i;
	}

	int* devA1;
	int* devB1;
	int* devC1;

	hipMalloc((void**)&devA1, sizeof(int) * N);
	hipMalloc((void**)&devB1, sizeof(int) * N);
	hipMalloc((void**)&devC1, sizeof(int) * N);

	hipMemcpy(devA1, a1, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(devB1, b1, sizeof(int) * N, hipMemcpyHostToDevice);

	Blockadd << <N, 1 >> > (devA1, devB1, devC1);

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(c1, devC1, sizeof(int) * N, hipMemcpyDeviceToHost);

	for (int i = 0; i<N; i++)
	{
		printf("%d + %d = %d\n", a1[i], b1[i], c1[i]);
	}

	hipEventDestroy(syncEvent);

	hipFree(devA1);
	hipFree(devB1);
	hipFree(devC1);

	/////////////////////////////////////////
	int a2[N], b2[N], c2[N];

	for (int i = 0; i<N; i++)
	{
		a2[i] = i*i;
		b2[i] = i * i*i;	
	}

	int* devA2;
	int* devB2;
	int* devC2;

	hipMalloc((void**)&devA2, sizeof(int) * N);
	hipMalloc((void**)&devB2, sizeof(int) * N);
	hipMalloc((void**)&devC2, sizeof(int) * N);

	hipMemcpy(devA2, a2, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(devB2, b2, sizeof(int) * N, hipMemcpyHostToDevice);

	Threadadd << <1, N >> > (devA2, devB2, devC2);

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(c2, devC2, sizeof(int) * N, hipMemcpyDeviceToHost);


	hipEventDestroy(syncEvent);

	hipFree(devA2);
	hipFree(devB2);
	hipFree(devC2);

	for (int i = 0; i<N; i++)
	{
		printf("%d + %d = %d\n", a2[i], b2[i], c2[i]);
	}

	std::system("pause");

	return 0;
}