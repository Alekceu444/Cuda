﻿
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>


#define N   10
#define M   5

__global__ void add(int *a, int *b, int *c)
{
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	c[tid] = a[tid] + b[tid];
}

__host__ int main(void)
{
	int a[N*M], b[N*M], c[N*M];

	for (int i = 0; i<N*M; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	int* devA;
	int* devB;
	int* devC;

	hipMalloc((void**)&devA, sizeof(int) * N*M);
	hipMalloc((void**)&devB, sizeof(int) * N*M);
	hipMalloc((void**)&devC, sizeof(int) * N*M);

	hipMemcpy(devA, a, sizeof(int) * N*M, hipMemcpyHostToDevice);
	hipMemcpy(devB, b, sizeof(int) * N*M, hipMemcpyHostToDevice);

	//printf("%d + %d \n", (N + (M - 1) / M),M);
	add <<<(N+(M-1)/M),M>>> (devA, devB, devC);
	//add <<<(N, M >>>(devA, devB, devC);

	hipEvent_t syncEvent;

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(c, devC, sizeof(int) * N*M, hipMemcpyDeviceToHost);

	for (int i = 0; i<N*M; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipEventDestroy(syncEvent);

	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	
	std::system("pause");

	return 0;
}