
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include <ctime>
#include <cstdlib>

#define NUM_BINS	256
#define N			9192
#define NUM_THREADS 512


__global__ void histogram(int * histogramm, int * arrays)
{

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int num = arrays[tid];
	histogramm[num] += 1;
}


int main(void) {

	srand(time(NULL));

	int a[N], b[NUM_BINS];


	for (int i = 0; i < N; i++) {

		a[i] = rand()%256;
	
	}

	for (int i = 0; i < NUM_BINS; i++) {

		b[i] = 0;

	}

	int* devA;
	int* devB;

	hipMalloc((void**)&devA, sizeof(int) * N);
	hipMalloc((void**)&devB, sizeof(int) * NUM_BINS);

	hipMemcpy(devA, a, sizeof(int) * N, hipMemcpyHostToDevice);
	

	histogram <<< ( N / NUM_THREADS), NUM_THREADS >>>(devB, devA);


	hipEvent_t syncEvent;

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(b, devB, sizeof(int) * NUM_BINS, hipMemcpyDeviceToHost);


	for (int i = 0; i < NUM_BINS; i++)
		printf("%d :: %d\n", i, b[i]);

	hipEventDestroy(syncEvent);

	hipFree(devA);
	hipFree(devB);

	std::system("pause");

	return 0;

}