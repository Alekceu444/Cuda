#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include ""

#define N   2
#define M   10

__global__ void summ(int *a, int *b, int *c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int tid = i * M + j;
	c[tid] = a[tid] + b[tid];

}

__host__ int main(void)
{
	int a[M*M], b[M*M], c[M*M];

	for (int i = 0; i<M*M; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}
	
	for (int i = 0; i<M; i++)
	{
		for (int j = 0; j < M; j++) {
			printf("%d ", a[i*M + j]);
		}
		printf("\n");
	}

	printf("-------------------------------- \n");

	for (int i = 0; i<M; i++)
	{
		for (int j = 0; j < M; j++) {
			printf("%d ", b[i*M + j]);
		}
		printf("\n");
	}

	printf("-------------------------------- \n");

	int* devA;
	int* devB;
	int* devC;

	hipMalloc((void**)&devA, sizeof(int) * M*M);
	hipMalloc((void**)&devB, sizeof(int) * M*M);
	hipMalloc((void**)&devC, sizeof(int) * M*M);

	hipMemcpy(devA, a, sizeof(int) * M*M, hipMemcpyHostToDevice);
	hipMemcpy(devB, b, sizeof(int) * M*M, hipMemcpyHostToDevice);

	dim3 blocks(M / N, M / N);
	dim3 threads(N, N);

	summ << <blocks, threads>> > (devA, devB, devC);

	hipEvent_t syncEvent;

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(c, devC, sizeof(int) * M*M, hipMemcpyDeviceToHost);

	for (int i = 0; i<M; i++)
	{
		for (int j = 0; j < M; j++) {
			printf("%d ", c[i*M+j]);
		}
		printf("\n");
	}

	hipEventDestroy(syncEvent);

	hipFree(devA);
	hipFree(devB);
	hipFree(devC);

	std::system("pause");

	return 0;
}